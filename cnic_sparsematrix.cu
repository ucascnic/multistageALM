#include "hip/hip_runtime.h"
#include"cnic_sparsematrix.h"
#include<iostream>
#include<hipsparse.h>
#include"matrix_function.h"
#include<assert.h>
#include<thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include<hipblas.h>
#define ARMA_ALLOW_FAKE_GCC
#include<armadillo>

static hipsparseHandle_t cusparse_handle = 0;
hipsparseStatus_t Status_t = hipsparseCreate(&cusparse_handle);
static void init_cusparse() {
  if (cusparse_handle == 0) {
    if (Status_t != HIPSPARSE_STATUS_SUCCESS) {
      printf("CUSPARSE Library initialization failed");
    }
  }
}
COOMatrix::COOMatrix(int m, int n,int n_ele){


    this->n_cols = n;
    this->n_rows = m;
    this->n_element = n_ele;
    // cpudata

    this->data = std::vector<double>(n_ele,0.0);
    this->col = std::vector<int>(n_ele,0);
    this->row = std::vector<int>(n_ele,0);
    //gpu data
    this->cudata = this->data;
    this->curow = this->row;
    this->cucol = this->col;


}
COOMatrix::COOMatrix(sp_mat & inputmat){

    //uvec res =  find(  inputmat );

    sp_mat::const_iterator it     = inputmat.begin();
    sp_mat::const_iterator it_end = inputmat.end();
    int n = inputmat.n_nonzero;
    this->n_element = n;
    this->data = std::vector<double>(n,0.0);
    this->col = std::vector<int>(n,0);
    this->row = std::vector<int>(n,0);
    this->n_rows = inputmat.n_rows;
    this->n_cols = inputmat.n_cols;


    int i = 0;
    for( ; it != it_end; ++it)
      {
       this->data[i] =  (*it) ;
       this->row[i] =  it.row();
        this->col[i] =  it.col();
        i++;
      }


    this->cudata = this->data;
    this->curow = this->row;
    this->cucol = this->col;

}




void CSRMatrix::print_matrix(){

//    for (int i = 0 ; i < this->n_element; ++i){
//        printf("%.4f\n",this->cucol[i]);
//    }
    show_res_T<double>((double*)thrust::raw_pointer_cast(this->cudata.data()),this->n_element);
}

void sortcoo_and_to_csr(int rows,int cols,int N,
                        double *val, int * row_ptrl, int *col_ptrl,
                        int *crs_row_ptrl){



    size_t workspace_size = 0;
    Status_t = hipsparseXcoosort_bufferSizeExt(
        cusparse_handle,
        rows, cols,
        N,
        row_ptrl,
        col_ptrl,
        &workspace_size);
    assert( HIPSPARSE_STATUS_SUCCESS == Status_t);
    double * buffer_for_coo_sort;
    CHECK(hipMalloc(&buffer_for_coo_sort, sizeof(char) * workspace_size));
    int * indptrl_cu;
    CHECK(hipMalloc(&indptrl_cu, sizeof(int) * N));
    int * indptrl_cu_mid = (int*) malloc( sizeof(int) *N);
    for ( int i =  0 ;i  < N ; ++i) indptrl_cu_mid[i] = i;


    hipMemcpy(indptrl_cu,indptrl_cu_mid,sizeof(int) * N,hipMemcpyHostToDevice);
    Status_t = hipsparseXcoosortByRow(
        cusparse_handle,
        rows, cols,
        N,
        row_ptrl,
        col_ptrl,
        indptrl_cu,
        buffer_for_coo_sort);


   Status_t = hipsparseDgthr(cusparse_handle,
                          N,
                          val,
                          buffer_for_coo_sort,
                          indptrl_cu,
                          HIPSPARSE_INDEX_BASE_ZERO);



   hipMemcpy(val,buffer_for_coo_sort,
              N*sizeof(double),hipMemcpyDeviceToDevice);
   assert( HIPSPARSE_STATUS_SUCCESS == Status_t);

    Status_t = hipsparseXcoo2csr(cusparse_handle,
        row_ptrl, N, rows,
        crs_row_ptrl, HIPSPARSE_INDEX_BASE_ZERO);
    assert( HIPSPARSE_STATUS_SUCCESS == Status_t);

    hipFree(buffer_for_coo_sort);
    hipFree(indptrl_cu);




}
void CSRMatrix::create_matrix_from_coo(COOMatrix * coo_matrix){

    int n = coo_matrix->n_element;
    this->cudata = coo_matrix->cudata;
    this->cucol = coo_matrix->cucol;
    this->csr_row_ptrl  = thrust::device_vector<int>(coo_matrix->n_rows + 1);


    sortcoo_and_to_csr(coo_matrix->n_rows,coo_matrix->n_cols,n,
                       thrust::raw_pointer_cast(this->cudata.data()),
                       thrust::raw_pointer_cast(coo_matrix->curow.data()),
                       thrust::raw_pointer_cast(this->cucol.data()),
                       thrust::raw_pointer_cast(this->csr_row_ptrl.data()));


}
CSRMatrix::CSRMatrix(int size,int rows,int cols){
    this->n_element = size;
    this->n_rows = rows;
    this->n_cols = cols;

}

cuVec::cuVec(thrust::host_vector<double> &data){
    this->n_element = data.size();
    this->cudata = data;
}














#include<hip/hip_runtime_api.h>
#define WARP_ 32
inline __device__  double __shfl_down_(double var, unsigned int srcLane, int width=WARP_) {
  int2 a = *reinterpret_cast<int2*>(&var);
  a.x = __shfl_down(a.x, srcLane, width);
  a.y = __shfl_down(a.y, srcLane, width);
  return *reinterpret_cast<double*>(&a);
}
__global__ void kernal_mat_u_32_wrap(double *kernel_cudata,
                                     int *csr_row_ptrl,int *cucol,
                                     double *u_cudata,
                                     double *output_cudata,int n_rows);
//void sp_matrix_times_V(CSRMatrix*A,cuVec*b,cuVec *c){
//    //std::cout << A->n_rows << std::endl;
//    dim3 block(A->n_rows);
//    c->cudata.resize(A->n_rows);
//    kernal_mat_u_32_wrap<<<block,WARP_>>>(thrust::raw_pointer_cast(A->cudata.data()),
//                                          thrust::raw_pointer_cast(A->csr_row_ptrl.data()),
//                                          thrust::raw_pointer_cast(A->cucol.data()),
//                                          thrust::raw_pointer_cast(b->cudata.data()),
//                                          thrust::raw_pointer_cast(c->cudata.data()),
//                                          A->n_rows);
//    c->n_element = A->n_rows;

//}
void sp_matrix_times_V(CSRMatrix*A,cuVec*b,cuVec *c){


    int base, nnz;
    int nnzb;
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    hipsparseDirection_t dirA = HIPSPARSE_DIRECTION_COLUMN;
    int m = A->n_rows;
    int n = A->n_cols;
    int blockDim = 2;
    int *bsrRowPtrC;
    hipsparseMatDescr_t descrA = NULL;
    hipsparseMatDescr_t descrC = NULL;
    hipsparseCreateMatDescr(&descrA);
    hipsparseCreateMatDescr(&descrC);

    int mb = (m + blockDim-1)/blockDim;
    int nb = (n + blockDim-1)/blockDim;
    hipMalloc((void**)&bsrRowPtrC, sizeof(int) *(mb+1));
    hipsparseXcsr2bsrNnz(handle, dirA, m, n,
     descrA, thrust::raw_pointer_cast(A->csr_row_ptrl.data()),
             thrust::raw_pointer_cast(A->cucol.data()), blockDim,
            descrC, bsrRowPtrC, &nnzb);

    int*bsrColIndC;
    double*bsrValC;
    hipMalloc((void**)&bsrColIndC, sizeof(int)*nnzb);
    hipMalloc((void**)&bsrValC, sizeof(double)*(blockDim*blockDim)*nnzb);
    hipsparseDcsr2bsr(handle, dirA, m, n,
     descrA, thrust::raw_pointer_cast(A->cudata.data()), thrust::raw_pointer_cast(A->csr_row_ptrl.data()),
                     thrust::raw_pointer_cast(A->cucol.data()), blockDim,
     descrC, bsrValC, bsrRowPtrC, bsrColIndC);
    // step 2: allocate vector x and vector y large enough for bsrmv
    double *x;
    double *y;
    hipMalloc((void**)&x, sizeof(double)*(nb*blockDim));
    hipMalloc((void**)&y, sizeof(double)*(mb*blockDim));
    hipMemcpy(x, thrust::raw_pointer_cast(b->cudata.data()), sizeof(double)*n, hipMemcpyDeviceToDevice);
    hipMemcpy(y, thrust::raw_pointer_cast(c->cudata.data()), sizeof(double)*m, hipMemcpyDeviceToDevice);
    // step 3: perform bsrmv
    double alpha = 1.;
    double beta = 0.;
    hipsparseDbsrmv(handle, dirA, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                   mb, nb, nnzb, &alpha,
     descrC, bsrValC, bsrRowPtrC, bsrColIndC, blockDim, x, &beta, y);

    hipMemcpy(thrust::raw_pointer_cast(c->cudata.data()),y, sizeof(double)*m, hipMemcpyDeviceToDevice);



}
void sp_matrix_times_V_ptrl(CSRMatrix*A,double*b,double *c){

    dim3 block(A->n_rows);
    kernal_mat_u_32_wrap<<<block,WARP_>>>(thrust::raw_pointer_cast(A->cudata.data()),
                                          thrust::raw_pointer_cast(A->csr_row_ptrl.data()),
                                          thrust::raw_pointer_cast(A->cucol.data()),
                                          b,
                                          c,
                                          A->n_rows);


}
BSRMatrix::BSRMatrix(CSRMatrix *A){


    int nnzb_;
    hipsparseCreate(&this->handle);
    dirA = HIPSPARSE_DIRECTION_COLUMN;
    m_rows = A->n_rows;
    n_cols = A->n_cols;
    blockDim = 2;

    descrA = NULL;
    descrC = NULL;
    hipsparseCreateMatDescr(&this->descrA);
    hipsparseCreateMatDescr(&this->descrC);

    int mb = (m_rows + blockDim-1)/blockDim;

    this->bsrRowPtrC.reserve(mb+1);
    //hipMalloc((void**)&bsrRowPtrC, sizeof(int) *(mb+1));
    hipsparseXcsr2bsrNnz(handle, dirA, m_rows, n_cols,
     descrA, thrust::raw_pointer_cast(A->csr_row_ptrl.data()),
             thrust::raw_pointer_cast(A->cucol.data()), blockDim,
            descrC, thrust::raw_pointer_cast(this->bsrRowPtrC.data()), &nnzb_);

    bsrColIndC.reserve(nnzb_);
    bsrValC.reserve((blockDim*blockDim)*nnzb_);
    //hipMalloc((void**)&bsrColIndC, sizeof(int)*nnzb);
    //hipMalloc((void**)&bsrValC, sizeof(double)*(blockDim*blockDim)*nnzb);
    hipsparseDcsr2bsr(handle, dirA, m_rows, n_cols,
     descrA, thrust::raw_pointer_cast(A->cudata.data()),
                     thrust::raw_pointer_cast(A->csr_row_ptrl.data()),
                     thrust::raw_pointer_cast(A->cucol.data()), blockDim,
                     descrC, thrust::raw_pointer_cast(bsrValC.data()),
                     thrust::raw_pointer_cast(bsrRowPtrC.data()),
                     thrust::raw_pointer_cast(bsrColIndC.data()));
    this->nnzb = nnzb_;

}
void sp_bsr_matrix_times_V_ptrl(BSRMatrix*A,double*b,double *c){

    int blockDim = A->blockDim;
    int m_rows = A->m_rows;
    int n_cols = A->n_cols;
    int mb = (m_rows + blockDim-1)/blockDim;
    int nb = (n_cols + blockDim-1)/blockDim;

    // step 2: allocate vector x and vector y large enough for bsrmv
    double *x;
    double *y;
    hipMalloc((void**)&x, sizeof(double)*(nb*blockDim));
    hipMalloc((void**)&y, sizeof(double)*(mb*blockDim));
    hipMemcpy(x, b, sizeof(double)*n_cols, hipMemcpyDeviceToDevice);
    hipMemcpy(y, c, sizeof(double)*m_rows, hipMemcpyDeviceToDevice);
    // step 3: perform bsrmv
    double alpha = 1.;
    double beta = 0.;
    hipsparseDbsrmv(A->handle, A->dirA, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                   mb, nb, A->nnzb, &alpha,
     A->descrC, thrust::raw_pointer_cast(A->bsrValC.data()),
                   thrust::raw_pointer_cast(A->bsrRowPtrC.data()),
                   thrust::raw_pointer_cast(A->bsrColIndC.data()),
                   blockDim, x, &beta, y);
    hipMemcpy(c,y, sizeof(double)*m_rows, hipMemcpyDeviceToDevice);
    hipFree(x);
    hipFree(y);


}
/*
if((row >= 404)&&(lane_id == 0)){
    printf("start ind =%d,end ind = %d\n",begin_index,end_index);
    for (int i  = 0 ; i < n_rows+1; ++i){
        printf("%d\t",csr_row_ptrl[i]);
    }
for(int i = begin_index + lane_id; i < end_index; i+=WARP_){
    if ( kernel_cudata[i] > 0){
        printf("%.2f\n",kernel_cudata[i]);
    }

}
}*/
__global__ void kernal_mat_u_32_wrap(double *kernel_cudata,
                                     int *csr_row_ptrl,int *cucol,
                                     double *u_cudata,
                                     double *output_cudata,int n_rows){

    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int vector_id = thread_id / WARP_;
    int lane_id = thread_id % WARP_;

    int row = vector_id;



    if(row <  n_rows){
        int begin_index =  csr_row_ptrl[row];
        int end_index =  csr_row_ptrl[row+1];

        double thread_sum = 0.0;
        for(int i = begin_index + lane_id; i < end_index; i+=WARP_)
            thread_sum += kernel_cudata[i] *  u_cudata [ cucol[i]];

//        if(row == 2423){
//            printf("csr_row_ptrl=%d   %d \n",csr_row_ptrl[row],csr_row_ptrl[row+1]);
//        }


        int temp = WARP_/2;
        while(temp >= 1){
            thread_sum += __shfl_down_(thread_sum, temp);
            temp >>= 1;
        }

        if ( lane_id == 0) {
              output_cudata[row] =  thread_sum;
        }

    }



}



void cuVec::print_vec(){
    show_res_T<double>((double*)thrust::raw_pointer_cast(this->cudata.data()),this->n_element);

}

//matrix transpose
void matrixTrans(CSRMatrix*A,CSRMatrix*B){

    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    COOMatrix matrix_temp(A->n_cols,A->n_rows,A->n_element);
    matrix_temp.cudata = A->cudata;
    matrix_temp.curow =  A->cucol;
    hipsparseXcsr2coo(handle,
     thrust::raw_pointer_cast(A->csr_row_ptrl.data()),
     A->n_element,
     A->n_rows,
     thrust::raw_pointer_cast(matrix_temp.cucol.data()),
     HIPSPARSE_INDEX_BASE_ZERO);
    B->create_matrix_from_coo(&matrix_temp);


    hipsparseDestroy(handle);

}





