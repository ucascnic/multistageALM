#include "hip/hip_runtime.h"
#include"cnic_sparsematrix.h"
#include<iostream>
#include<hipsparse.h>
#include<hip/hip_runtime_api.h>
#include"matrix_function.h"
#include<assert.h>
#include<thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include<hipblas.h>
#define ARMA_ALLOW_FAKE_GCC
#include<armadillo>

static hipsparseHandle_t cusparse_handle = 0;
hipsparseStatus_t Status_t = hipsparseCreate(&cusparse_handle);
static void init_cusparse() {
  if (cusparse_handle == 0) {
    if (Status_t != HIPSPARSE_STATUS_SUCCESS) {
      printf("CUSPARSE Library initialization failed");
    }
  }
}
COOMatrix::COOMatrix(int m, int n,int n_ele){


    this->n_cols = n;
    this->n_rows = m;
    this->n_element = n_ele;
    // cpudata

    this->data = std::vector<double>(n_ele,0.0);
    this->col = std::vector<int>(n_ele,0);
    this->row = std::vector<int>(n_ele,0);
    //gpu data
    this->cudata = this->data;
    this->curow = this->row;
    this->cucol = this->col;


}
COOMatrix::COOMatrix(sp_mat & inputmat){

    //uvec res =  find(  inputmat );

    sp_mat::const_iterator it     = inputmat.begin();
    sp_mat::const_iterator it_end = inputmat.end();
    int n = inputmat.n_nonzero;
    this->n_element = n;
    this->data = std::vector<double>(n,0.0);
    this->col = std::vector<int>(n,0);
    this->row = std::vector<int>(n,0);
    this->n_rows = inputmat.n_rows;
    this->n_cols = inputmat.n_cols;


    int i = 0;
    for( ; it != it_end; ++it)
      {
       this->data[i] =  (*it) ;
       this->row[i] =  it.row();
        this->col[i] =  it.col();
        i++;
      }


    this->cudata = this->data;
    this->curow = this->row;
    this->cucol = this->col;

}




void CSRMatrix::print_matrix(){

//    for (int i = 0 ; i < this->n_element; ++i){
//        printf("%.4f\n",this->cucol[i]);
//    }
    show_res_T<double>((double*)thrust::raw_pointer_cast(this->cudata.data()),this->n_element);
}

void sortcoo_and_to_csr(int rows,int cols,int N,
                        double *val, int * row_ptrl, int *col_ptrl,
                        int *crs_row_ptrl){



    size_t workspace_size = 0;
    Status_t = hipsparseXcoosort_bufferSizeExt(
        cusparse_handle,
        rows, cols,
        N,
        row_ptrl,
        col_ptrl,
        &workspace_size);
    assert( HIPSPARSE_STATUS_SUCCESS == Status_t);
    double * buffer_for_coo_sort;
    CHECK(hipMalloc(&buffer_for_coo_sort, sizeof(char) * workspace_size));
    int * indptrl_cu;
    CHECK(hipMalloc(&indptrl_cu, sizeof(int) * N));
    int * indptrl_cu_mid = (int*) malloc( sizeof(int) *N);
    for ( int i =  0 ;i  < N ; ++i) indptrl_cu_mid[i] = i;


    hipMemcpy(indptrl_cu,indptrl_cu_mid,sizeof(int) * N,hipMemcpyHostToDevice);
    Status_t = hipsparseXcoosortByRow(
        cusparse_handle,
        rows, cols,
        N,
        row_ptrl,
        col_ptrl,
        indptrl_cu,
        buffer_for_coo_sort);


   Status_t = hipsparseDgthr(cusparse_handle,
                          N,
                          val,
                          buffer_for_coo_sort,
                          indptrl_cu,
                          HIPSPARSE_INDEX_BASE_ZERO);



   hipMemcpy(val,buffer_for_coo_sort,
              N*sizeof(double),hipMemcpyDeviceToDevice);
   assert( HIPSPARSE_STATUS_SUCCESS == Status_t);

    Status_t = hipsparseXcoo2csr(cusparse_handle,
        row_ptrl, N, rows,
        crs_row_ptrl, HIPSPARSE_INDEX_BASE_ZERO);
    assert( HIPSPARSE_STATUS_SUCCESS == Status_t);

    hipFree(buffer_for_coo_sort);
    hipFree(indptrl_cu);




}
#include"cnicsparsematrix.h"
void CSRMatrix::create_matrix_from_coo(COOMatrix * coo_matrix){

    int n = coo_matrix->n_element;
    this->cudata = coo_matrix->cudata;
    this->cucol = coo_matrix->cucol;
    this->csr_row_ptrl  = thrust::device_vector<int>(coo_matrix->n_rows + 1);



    COORowPtrl coorowptrl;
    coorowptrl.ptr.resize(n);
    for (int i =0;i<n;++i){
        coorowptrl.ptr[i] = coo_matrix->row[i];
    }


    CSRIntMatrix  csr;
    csr.col.resize(this->cucol.size());
    csr.ptr.resize(coo_matrix->n_rows+1);
    for (int i = 0 ;i < this->cucol.size();++i){
        csr.col[i]=coo_matrix->col[i];
    }


    ptrdiff_t nnz = n;
    csr.val.resize(nnz);
    for (int i = 0 ; i < nnz;++i){
        csr.val[i] = i;
    }


    qsortCOO2CSR<int>(coorowptrl.ptr.data(), csr.col.data(), csr.val.data(), 0, nnz - 1);

    compressIndices(coorowptrl.ptr.data(), csr.ptr.data(), coo_matrix->n_rows, nnz);


    thrust::host_vector<int> temp1(csr.ptr.size());

    for (int i = 0 ; i< csr.ptr.size();++i){
        temp1[i] = csr.ptr[i];

    }


    this->csr_row_ptrl = temp1;

    thrust::host_vector<double> temp2(csr.col.size());

    for (int i = 0 ; i< csr.col.size();++i){
        temp2[i] = csr.col[i];
    }
    this->cucol = temp2;


    thrust::host_vector<double> temp3 = coo_matrix->data;
    thrust::host_vector<double> temp4 = coo_matrix->data;
    for (int i = 0 ; i< temp3.size();++i){
        temp4[i] = temp3[csr.val[i]];
    }
    this->cudata = temp4;
    show_res_T<double>((double*)thrust::raw_pointer_cast(this->cudata.data()),10);

    return   ;

}
CSRMatrix::CSRMatrix(int size,int rows,int cols){
    this->n_element = size;
    this->n_rows = rows;
    this->n_cols = cols;

}

cuVec::cuVec(thrust::host_vector<double> &data){
    this->n_element = data.size();
    this->cudata = data;
}














#include<hip/hip_runtime_api.h>
#define WARP_ 32
inline __device__  double __shfl_down_(double var, unsigned int srcLane, int width=WARP_) {
  int2 a = *reinterpret_cast<int2*>(&var);
  a.x = __shfl_down(a.x, srcLane, width);
  a.y = __shfl_down(a.y, srcLane, width);
  return *reinterpret_cast<double*>(&a);
}
__global__ void kernal_mat_u_32_wrap(double *kernel_cudata,
                                     int *csr_row_ptrl,int *cucol,
                                     double *u_cudata,
                                     double *output_cudata,int n_rows);
//void sp_matrix_times_V(CSRMatrix*A,cuVec*b,cuVec *c){
//    //std::cout << A->n_rows << std::endl;
//    dim3 block(A->n_rows);
//    c->cudata.resize(A->n_rows);
//    kernal_mat_u_32_wrap<<<block,WARP_>>>(thrust::raw_pointer_cast(A->cudata.data()),
//                                          thrust::raw_pointer_cast(A->csr_row_ptrl.data()),
//                                          thrust::raw_pointer_cast(A->cucol.data()),
//                                          thrust::raw_pointer_cast(b->cudata.data()),
//                                          thrust::raw_pointer_cast(c->cudata.data()),
//                                          A->n_rows);
//    c->n_element = A->n_rows;

//}

void sp_matrix_times_V_ptrl(CSRMatrix*A,double*b,double *c){

    dim3 block(A->n_rows);
    kernal_mat_u_32_wrap<<<block,WARP_>>>(thrust::raw_pointer_cast(A->cudata.data()),
                                          thrust::raw_pointer_cast(A->csr_row_ptrl.data()),
                                          thrust::raw_pointer_cast(A->cucol.data()),
                                          b,
                                          c,
                                          A->n_rows);


}


/*
if((row >= 404)&&(lane_id == 0)){
    printf("start ind =%d,end ind = %d\n",begin_index,end_index);
    for (int i  = 0 ; i < n_rows+1; ++i){
        printf("%d\t",csr_row_ptrl[i]);
    }
for(int i = begin_index + lane_id; i < end_index; i+=WARP_){
    if ( kernel_cudata[i] > 0){
        printf("%.2f\n",kernel_cudata[i]);
    }

}
}*/
__global__ void kernal_mat_u_32_wrap(double *kernel_cudata,
                                     int *csr_row_ptrl,int *cucol,
                                     double *u_cudata,
                                     double *output_cudata,int n_rows){

    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int vector_id = thread_id / WARP_;
    int lane_id = thread_id % WARP_;

    int row = vector_id;



    if(row <  n_rows){
        int begin_index =  csr_row_ptrl[row];
        int end_index =  csr_row_ptrl[row+1];

        double thread_sum = 0.0;
        for(int i = begin_index + lane_id; i < end_index; i+=WARP_)
            thread_sum += kernel_cudata[i] *  u_cudata [ cucol[i]];

//        if(row == 2423){
//            printf("csr_row_ptrl=%d   %d \n",csr_row_ptrl[row],csr_row_ptrl[row+1]);
//        }


        int temp = WARP_/2;
        while(temp >= 1){
            thread_sum += __shfl_down_(thread_sum, temp);
            temp >>= 1;
        }

        if ( lane_id == 0) {
              output_cudata[row] =  thread_sum;
        }

    }



}



void cuVec::print_vec(){
    show_res_T<double>((double*)thrust::raw_pointer_cast(this->cudata.data()),this->n_element);

}

//matrix transpose
void matrixTrans(CSRMatrix*A,CSRMatrix*B){

    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    COOMatrix matrix_temp(A->n_cols,A->n_rows,A->n_element);
    matrix_temp.cudata = A->cudata;
    matrix_temp.curow =  A->cucol;
    hipsparseXcsr2coo(handle,
     thrust::raw_pointer_cast(A->csr_row_ptrl.data()),
     A->n_element,
     A->n_rows,
     thrust::raw_pointer_cast(matrix_temp.cucol.data()),
     HIPSPARSE_INDEX_BASE_ZERO);
    B->create_matrix_from_coo(&matrix_temp);


    hipsparseDestroy(handle);

}





