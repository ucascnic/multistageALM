#include"matrix_function.h"
#include<hip/hip_runtime_api.h>
#include <hipblas.h>

void matrix_times(hipblasHandle_t handle,double *f,double *g,int mf,int nf, int kg,double *res){

    hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,mf,kg,nf,
                 &alpha1 ,f ,mf ,g ,nf ,&beta,res,mf);
}

void matrix_timesV(hipblasHandle_t handle,double *f,double *g,int mf,int nf,double *res){

    hipblasDgemv(handle, HIPBLAS_OP_N, mf, nf, &alpha1,
                f, mf, g, 1, &beta, res,1);
}
void matrixT_timesV(hipblasHandle_t handle,double *f,double *g,int mf,int nf,double *res){

    hipblasDgemv(handle, HIPBLAS_OP_T, mf, nf, &alpha1,
                f, mf, g, 1, &beta, res,1);
}
void matrix_timesT(hipblasHandle_t handle,double *f,double *g,int mf,int nf, int kg,double *res){

    hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,mf,kg,nf,
                 &alpha1 ,f,mf,g,kg,&beta,res,mf);
}

void matrix_AtimesAT(hipblasHandle_t handle,double *f,int mf,int nf,double *res){

    hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,mf,mf,nf,
                 &alpha1 ,f,mf,f,mf,&beta,res,mf);
}
void matrix_ATtimesA(hipblasHandle_t handle,double *f,int mf,int nf,double *res){

    hipblasDgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,nf,nf,mf,
                 &alpha1 ,f,mf,f,mf,&beta,res,nf);
}
void matrixT(hipblasHandle_t handle, double *P, double *Pt, int m, int n){
    hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, &alpha1, P, m, &beta,
                Pt,n, Pt, n);

}


void show_res(double *s,int n){
    double *res = (double *)malloc(n*sizeof(double));
    CHECK(hipMemcpy(res,s,n*sizeof(double),hipMemcpyDeviceToHost));
    for (int i = 0 ; i< n;++i){
        printf("%.6f\t",res[i]);
    }

    printf("\n");
    free(res);
}





