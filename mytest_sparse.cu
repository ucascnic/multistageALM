#include"mytest_sparse.h"
#include<string>
#include<vector>
#include<hip/hip_runtime_api.h>
#include<cnic_sparsematrix.h>
#include<matrix_function.h>
#include<thrust/device_ptr.h>
#include<thrust/copy.h>
#include"mytest_sparse.h"


void run_test_for_sparse_mv()
{
    for (int n = 1000; n < 1000000; n += 10){

        sp_mat B;
        std::cout << "run test for size = " << n  << std::endl;
        std::string file = "/home/chenyidong/multiblock/test_sparse/"+std::to_string(n)+".txt";
        B.load(file.data(),coord_ascii);


        mat  test_vec  = randn(n,1);

        thrust::host_vector<double>  x(test_vec.memptr(),test_vec.memptr()+n);


        test_vec =  B * test_vec;
        //std::cout << test_vec << std::endl;
        COOMatrix   test(B);

        CSRMatrix    csrttest( test.n_element,test.n_rows,test.n_cols);
        csrttest.create_matrix_from_coo(&test);


        cuVec  cuvec(x);



        cuVec  output(x);

        //output.print_vec();
        //sp_matrix_times_V(&csrttest,&cuvec,&output);
        BSRMatrix tt(&csrttest);
        sp_bsr_matrix_times_V_ptrl(&tt,
                                   thrust::raw_pointer_cast(cuvec.cudata.data()),
                                   thrust::raw_pointer_cast(output.cudata.data()));
        //output.print_vec();
        int result = check<double>(thrust::raw_pointer_cast(output.cudata.data()),test_vec.memptr(),test_vec.size());

        std::cout << result << std::endl;
        if (result != 1){
            std::cout << "error" << std::endl;
            exit(0);
        }
    }

}
void run_test_for_matrix_transpose(sp_mat &B){


    //int n = 100;
    //sp_mat B;
    //std::string file = "/home/chenyidong/multiblock/test_sparse/"+std::to_string(n)+".txt";
    //B.load(file.data(),coord_ascii);
    int n = B.n_rows;


    mat  test_vec  = ones(n,1);

    thrust::host_vector<double>  x(test_vec.memptr(),test_vec.memptr()+n);



    COOMatrix   test(B);
    CSRMatrix    csrttest( test.n_element,test.n_rows,test.n_cols);
    csrttest.create_matrix_from_coo(&test);

    CSRMatrix    csrttestT( test.n_element,test.n_cols,test.n_rows);
    matrixTrans(&csrttest,&csrttestT);
    //sp_mat BB = trans(B);
    //COOMatrix   test2(BB);
    //csrttestT.create_matrix_from_coo(&test2);

    cuVec  cuvec(x);
    cuVec  output(x);
    //output.print_vec();
    sp_matrix_times_V(&csrttestT,&cuvec,&output);



    test_vec =  trans(B) * test_vec;
    std::cout<< test_vec << std::endl;
    output.print_vec();



    int result = check<double>(thrust::raw_pointer_cast(output.cudata.data()),test_vec.memptr(),test_vec.size());
    std::cout << result << std::endl;
    if (result != 1){
        std::cout << "error" << std::endl;
        exit(0);
    }


}
